#include "hip/hip_runtime.h"
/*
 * main.cu
 *
 *  Created on: Nov 30, 2015
 *      Author: john
 */


#include "Bitmap.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>
#include <assert.h>

using namespace std;

const int BINS = 256;
const int BINS4ALL = BINS*16; // Using half warp size since I need 3 sets, and these bins need to fit in 48kb shared memory.

void CPU_histogram (unsigned char *in_red, unsigned char *in_blue, unsigned char *in_green, int N, int *h_red, int *h_blue, int *h_green, int bins)
{
  int i;
  // initialize histogram counts
  for (i = 0; i < bins; i++) {
	  h_red[i] = 0;
  	  h_blue[i] = 0;
  	  h_green[i] = 0;
  }

  // accummulate counts
  for (i = 0; i < N; i++) {
	  h_red[in_red[i]]++;
	  h_blue[in_blue[i]]++;
	  h_green[in_green[i]]++;
  }

}

__device__
void write_shared(int *in, int* bank, int i) {
	int temp = in[i];
	int v = temp & 0xFF;
	int v2 = (temp >> 8) & 0xFF;
	int v3 = (temp >> 16) & 0xFF;
	int v4 = (temp >> 24) & 0xFF;
	atomicAdd (bank + (v << 4), 1);
	atomicAdd (bank + (v2 << 4), 1);
	atomicAdd (bank + (v3 << 4), 1);
	atomicAdd (bank + (v4 << 4), 1);
}

__global__
void GPU_histogramRGB_atomic (int *in_red, int *in_blue, int *in_green, int N, int *h_red, int *h_blue, int *h_green) {
	int gloID = blockIdx.x*blockDim.x + threadIdx.x;
	int locID = threadIdx.x;
	int GRIDSIZE = gridDim.x*blockDim.x;

	__shared__ int localH_red[BINS4ALL];
	__shared__ int localH_blue[BINS4ALL];
	__shared__ int localH_green[BINS4ALL];

	int bankID = locID & 0x0F;
	int i;

	// initialize the local shared-memory bins
	for (i = locID; i < BINS4ALL; i += blockDim.x) {
		localH_red[i] = 0;
		localH_blue[i] = 0;
		localH_green[i] = 0;
	}

	__syncthreads();

	int *mySharedBank_red = localH_red + bankID;
	int *mySharedBank_blue = localH_blue + bankID;
	int *mySharedBank_green = localH_green + bankID;

	for (i = gloID; i < N; i += GRIDSIZE) {
		write_shared(in_red, mySharedBank_red, i);
		write_shared(in_blue, mySharedBank_blue, i);
		write_shared(in_green, mySharedBank_green, i);
	}

	__syncthreads ();


	for (i = locID; i < BINS4ALL; i += blockDim.x) {
		atomicAdd (h_red + (i >> 4), localH_red[i]);
		atomicAdd (h_blue + (i >> 4), localH_blue[i]);
		atomicAdd (h_green + (i >> 4), localH_green[i]);
	}

}


int main (int argc, char **argv) {
	Bitmap* bmp = new Bitmap("CAT2.bmp");

	int *d_in_red, *d_in_blue, *d_in_green;
	int *h_in_red, *h_in_blue, *h_in_green;
	int *cpu_hist_red, *cpu_hist_blue, *cpu_hist_green;
	int *d_hist_red, *d_hist_blue, *d_hist_green;
	int *hist_red, *hist_blue, *hist_green;

	int bins, N;

	h_in_red = (int *) bmp->pixels_red;
	h_in_blue = (int *) bmp->pixels_blue;
	h_in_green = (int *) bmp->pixels_green;
	N = ceil((bmp->x_dim * bmp->y_dim) / 4.0);

	bins = 256;

	hist_red = (int *) malloc (bins * sizeof (int));
	hist_blue = (int *) malloc (bins * sizeof (int));
	hist_green = (int *) malloc (bins * sizeof (int));

	cpu_hist_red = (int *) malloc (bins * sizeof (int));
	cpu_hist_blue = (int *) malloc (bins * sizeof (int));
	cpu_hist_green = (int *) malloc (bins * sizeof (int));

	CPU_histogram(bmp->pixels_red, bmp->pixels_blue, bmp->pixels_green, bmp->x_dim*bmp->y_dim, cpu_hist_red, cpu_hist_blue, cpu_hist_green, bins);



    // allocate and copy
    hipMalloc ((void **) &d_in_red, sizeof (int) * N);
    hipMalloc ((void **) &d_hist_red, sizeof (int) * bins);
    hipMemcpy (d_in_red, h_in_red, sizeof (int) * N, hipMemcpyHostToDevice);
    hipMemset (d_hist_red, 0, bins * sizeof (int));

    hipMalloc ((void **) &d_in_blue, sizeof (int) * N);
    hipMalloc ((void **) &d_hist_blue, sizeof (int) * bins);
    hipMemcpy (d_in_blue, h_in_blue, sizeof (int) * N, hipMemcpyHostToDevice);
    hipMemset (d_hist_blue, 0, bins * sizeof (int));

    hipMalloc ((void **) &d_in_green, sizeof (int) * N);
    hipMalloc ((void **) &d_hist_green, sizeof (int) * bins);
    hipMemcpy (d_in_green, h_in_green, sizeof (int) * N, hipMemcpyHostToDevice);
    hipMemset (d_hist_green, 0, bins * sizeof (int));

    // initialize two events
    hipStream_t str;
    hipEvent_t startT, endT;
    float duration;
    hipStreamCreate (&str);
    hipEventCreate (&startT);
    hipEventCreate (&endT);

    hipEventRecord (startT, str);
	GPU_histogramRGB_atomic <<<32, 1024, 0, str >>> (d_in_red, d_in_blue, d_in_green, N, d_hist_red, d_hist_blue, d_hist_green);

	hipEventRecord (endT, str);
	hipEventSynchronize (endT);

	hipMemcpy (hist_red, d_hist_red, sizeof (int) * bins, hipMemcpyDeviceToHost);
	hipMemcpy (hist_blue, d_hist_blue, sizeof (int) * bins, hipMemcpyDeviceToHost);
	hipMemcpy (hist_green, d_hist_green, sizeof (int) * bins, hipMemcpyDeviceToHost);
	hipEventElapsedTime (&duration, startT, endT);

	for (int i = 0; i < BINS; i++)
	    printf ("%i %i %i %i\n", i, hist_red[i], hist_blue[i], hist_green[i]);


	for (int i = 0; i < BINS; i++)
		if (cpu_hist_red[i] != hist_red[i] || cpu_hist_blue[i] != hist_blue[i] || cpu_hist_green[i] != hist_green[i])
			printf ("Calculation mismatch (static) at : %i\n", i);



	printf ("Kernel executed for %f ms\n", duration);

	hipStreamDestroy (str);
	hipEventDestroy (startT);
	hipEventDestroy (endT);

	hipFree ((void *) d_in_red);
	hipFree ((void *) d_hist_red);
	free (hist_red);
	hipFree ((void *) d_in_blue);
	hipFree ((void *) d_hist_blue);
	free (hist_blue);
	hipFree ((void *) d_in_green);
	hipFree ((void *) d_hist_green);
	free (hist_green);

	hipDeviceReset ();
	return 0;
}

